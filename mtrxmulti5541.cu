
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hiprand/hiprand.h>
#include<iostream>
#include<stdlib.h>
#include<time.h> 
#include<cstdio>
#include <assert.h>
#define M 500 
#define N 500
#define K 400
#define C 100
using namespace std;

__global__ void multi_kernel(int *mn,int *m, int *n){
  int xbidx = blockIdx.x;
  int ybidx = blockIdx.y;
  int tidx = threadIdx.x;
  __shared__ int sh_var[N];
  sh_var[tidx] = mn[N * ybidx + tidx] * m[K * tidx + xbidx];
  __syncthreads();

  n[K * ybidx + xbidx ] = 0;
  for(int i = 0; i<N; i++){
      n[K * ybidx + xbidx] = n[K * ybidx + xbidx] + sh_var[i];
      }
  }


int multiplication(){
  int *a,*b,*c;
  int an[M][N];
  int bn[N][K];
  int cn[M][K];
    
  //Generating random Matrix B
  for (int i = 0; i < N; i++){
      for (int  j = 0; j < K; j++){
          bn[i][j] = (int)rand() % 100 * sizeof(int);
      }
  }
  cout << "Matrix B generated" << endl;

  hipMallocManaged((void **)&b, N * K * sizeof(int));
  hipMemcpy(b, bn, N * K * sizeof(int), hipMemcpyHostToDevice);  
  dim3 gridDim(K,M);

  for (int i = 0; i < C; i++){
      for (int k = 0; k < M; k++){
          for (int l = 0; l < N; l++){
              an[k][l] = (int)rand() % 100 * sizeof(int);
              //printf("%d\n", &an[k][l]);
          }
      }
      hipMallocManaged((void **)&a, M * N * sizeof(int));
      hipMallocManaged((void **)&c, M * K * sizeof(int));
      hipMemcpy(a, an, M * N * sizeof(int), hipMemcpyHostToDevice);
      multi_kernel <<< gridDim, N >>> (a, b, c);
      hipMemcpy(cn, c, M * K * sizeof(int), hipMemcpyDeviceToHost);
      
      hipFree(a);
      hipFree(c);
  }
  hipFree(b);
  cout << "Completed Successfully" << endl;
  cout << "[" << M << "] " << "x" << " [" << N << "] " << "*"<< " [" << N << "] "<< "x" <<  " [" << K << "]"<< endl;
  return 0;  
} 

int main(){ 
  time_t start, end, t; 
  start = time(NULL);
 	srand((unsigned) time(&t));
   multiplication();  
  end = time(NULL); 
 // printf("%ld", &end);
  cout << "Total execution time: " << (end-start) << " seconds" << endl;
  return 0;
}